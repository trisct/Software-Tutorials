#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

int SHOW_ONLY = 15;

__global__ void vectorIncrease(float *A, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        A[i] += 1.;
    }
}

/**
 * Increase the values in the vector by 1
 * without error checks
 */
int main() {

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector incremental operation of %d elements]\n", numElements);

    // allocating memory and initialization
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
    }

    // Copy memory: host to device
    printf("Copy input data from the host memory to the CUDA device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024; // maximum 1024, otherwise none will be executed
    int blocksPerGrid = 50;
    vectorIncrease<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
    printf("CUDA kernel launched with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // Copy memory: device to host
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_B, d_A, size, hipMemcpyDeviceToHost);

    for (int i=0; i<min(SHOW_ONLY, numElements); i++) printf("%.5f ", h_A[i]); cout << "\n";
    for (int i=0; i<min(SHOW_ONLY, numElements); i++) printf("%.5f ", h_B[i]); cout << "\n";

    int correct_count = 0;
    for (int i=0; i<numElements; i++)
        if (h_B[i] == h_A[i] + 1)
            correct_count++;
    printf("%d out of %d are correct\n", correct_count, numElements);

    // Free memory
    hipFree(d_A);
    free(h_A);
    free(h_B);

    printf("Done\n");
    return 0;
}

